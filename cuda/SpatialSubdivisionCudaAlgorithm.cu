#include "hip/hip_runtime.h"
#include "Collision/CollisionAlgorithm.h"

#include <cmath>
#include <iostream>
#include <cstdint>
#include <vector>
#include <bitset>

#define MAX_BLOCK_SIZE 384 //Dividable by 32 so each block work with full wraps. 

// CUDA error checking macro.
#define CUDA_CHECK(error)                                                    \
    do {                                                                     \
        hipError_t err = error;                                             \
        if (err != hipSuccess) {                                            \
            std::cerr << "CUDA Error: " << hipGetErrorString(err)           \
                      << " at " << __FILE__ << ":" << __LINE__ << "\n";      \
            exit(EXIT_FAILURE);                                              \
        }                                                                    \
    } while (0)


struct object_id
{
    int particle_id {0}; ///< The unique ID of the particle.
    int home_cell {0};   ///< Flag indicating if the cell is the particle's home cell.
};


double* vecToArr(const std::vector<std::vector<double>>& vec) {
    std::size_t totalsize = 0;

    for (int i=0; i<vec.size(); i++) {
        totalsize += vec[i].size();
    }

    double* newarr=new double[totalsize];
    double* walkarr=newarr;

    for (int i=0; i<vec.size(); i++) {
        std::copy(vec[i].begin(), vec[i].end(), walkarr);
        walkarr += vec[i].size();
    }

    return newarr;
}

__device__ 
int64_t hash_coordinates(int64_t grid_x, int64_t grid_y, int64_t grid_z) {
    int64_t hash = 0;
    hash |= (grid_x & 0xFFFF);         // Place X in bits 0–15
    hash |= (grid_y & 0xFFFF) << 16;   // Place Y in bits 16–31
    hash |= (grid_z & 0xFFFF) << 32;   // Place Z in bits 32–47
    return hash;
}

__global__ 
void initializeObjectandCellArray(
    double *coordinates, 
    int64_t *cellIdArray, 
    object_id *objectIdArray, 
    int coordinatesLength, 
    double collisionDist) 
{
    double const cellLength {collisionDist * 1.5}; 
    double const collisionDistSq {collisionDist * collisionDist};

    int TIdx {threadIdx.x + blockDim.x * blockIdx.x};
    int coordIdx {3 * TIdx};
    int arrayIdx {8 * TIdx};

    // Finds the cell grid coordinate of the home cell.
    if  (TIdx < coordinatesLength) {
        int64_t grid_x = static_cast<int64_t>(floor(coordinates[coordIdx]     / cellLength));
        int64_t grid_y = static_cast<int64_t>(floor(coordinates[coordIdx + 1] / cellLength));
        int64_t grid_z = static_cast<int64_t>(floor(coordinates[coordIdx + 2] / cellLength));

        // Hashes and adds the home cell to cellIdArray list.
        cellIdArray[arrayIdx] = (hash_coordinates(grid_x, grid_y, grid_z));

        // Adds corresponding information regarding the particle to the ObjectIdArray.
        objectIdArray[arrayIdx] = object_id{TIdx, 1};

        // Checks if the particle is present in any neighbourign cells. Can be in maxiumum 7 more cells due to geometry
        int counter {1};
        for (int64_t dx = -1; dx <= 1; ++dx) {
            for (int64_t dy = -1; dy <= 1; ++dy) {
                for (int64_t dz = -1; dz <= 1; ++dz) {
                    if (dx == 0 && dy == 0 && dz == 0) {
                        continue;
                    }

                    double minX = (grid_x + dx) * cellLength;
                    double maxX = minX + cellLength;
                    double minY = (grid_y + dy) * cellLength;
                    double maxY = minY + cellLength;
                    double minZ = (grid_z + dz) * cellLength;
                    double maxZ = minZ + cellLength;

                    // Finding the nearest point of the relevant cell.
                    double nearestX = fmax(minX, fmin(coordinates[coordIdx], maxX));
                    double nearestY = fmax(minY, fmin(coordinates[coordIdx + 1], maxY));
                    double nearestZ = fmax(minZ, fmin(coordinates[coordIdx + 2], maxZ));

                    // Calculates distance to the relevant cell
                    double distSq =   (coordinates[coordIdx]     - nearestX) *     (coordinates[coordIdx] - nearestX)
                                    + (coordinates[coordIdx + 1] - nearestY) * (coordinates[coordIdx + 1] - nearestY)
                                    + (coordinates[coordIdx + 2] - nearestZ) * (coordinates[coordIdx + 2] - nearestZ);

                    // Adds the cell to the cellIdArray array if the distance between the particle and the relevant cell is less than the particle radius.
                    if (distSq <= collisionDistSq) {
                        cellIdArray[arrayIdx + counter] = (hash_coordinates(grid_x + dx, grid_y + dy, grid_z + dz));
                        objectIdArray[arrayIdx + counter] = object_id{TIdx, 0};
                        counter++;
                    }
                }
            }
        }
        for(; counter < 8; counter++) {
            cellIdArray[arrayIdx + counter] = 0xFFFFFFFFFFFF;  //Setting to a default value the particle doesn't cross into 7 other cells. 
            objectIdArray[arrayIdx + counter] = object_id{-1, 0};
        }
    }
}


__global__ 
void placeCellIdFlags(
    int64_t *d_cellIdArray, 
    int64_t *d_cellIdArrayTemp,
    object_id *d_objectIdArray, 
    object_id *d_objectIdArrayTemp, 
    int *d_flagCellIDStart,
    int *d_flagCellIDEnd,
    unsigned int arrayIdLength) 
{
    int TIdx = threadIdx.x + blockDim.x * blockIdx.x;

    // Shift the original d_cellIdArray one step Forwards.
    if (TIdx < (arrayIdLength - 1)) {
        d_cellIdArrayTemp[TIdx + 1] = d_cellIdArray[TIdx];
    }
    
    __syncthreads();

    // Check if the element on index i is not the same as the element on index i-1. If true place a start marker.
    if (TIdx < (arrayIdLength) && TIdx != 0) {
        d_flagCellIDStart[TIdx] = ((d_cellIdArrayTemp[TIdx] != d_cellIdArray[TIdx]) ? 1 : 0);        
    } else if  (TIdx == 0) {
        d_flagCellIDStart[TIdx] = 1;
    }

    __syncthreads();

    // Create a end marker array by shifting the start marker array on step to the left.
    if (TIdx < arrayIdLength && TIdx != 0) {
        d_flagCellIDEnd[TIdx - 1] = d_flagCellIDStart[TIdx];
    } else if (TIdx == arrayIdLength) {
        d_flagCellIDStart[TIdx] = 1;
    }
}


__device__ 
double calculateDistanceSq(
    double x1, 
    double y1, 
    double z1,
    double x2,
    double y2, 
    double z2) 
{
    double dx = x2 - x1;
    double dy = y2 - y1;
    double dz = z2 - z1;
    return dx * dx + dy * dy + dz * dz;
}

__device__ 
void checkCollision(
    int i, 
    int j,
    const double *coordinates,
    const object_id *objectIdArray,
    double collDistSq,
    int *counter) 
{
    // Only do distance checks if home_id == 1
    if (objectIdArray[i].home_cell == 1) {
        double p1 = objectIdArray[i].particle_id; 
        double p2 = objectIdArray[j].particle_id; 

        double distSq = calculateDistanceSq(
            coordinates[3 * p1], 
            coordinates[3 * p1 + 1], 
            coordinates[3 * p1 + 2], 
            coordinates[3 * p2], 
            coordinates[3 * p2 + 1], 
            coordinates[3 * p2 + 2]);

        if (distSq < collDistSq) {
            counter[i]++; 
        }
    }
}

__global__ 
void calculateNumberOfCollisions(
    const int64_t *d_cellIdArray,
    const object_id *d_objectIdArray,  
    const int *d_flagCellIdStart,
    const int *d_flagCellIdEnd,
    const double *coordinates,
    double collDistSq,
    unsigned int arrayIdLength,
    int *d_collisionCounter)
{
    // Thread index
    int TIdx = blockIdx.x * blockDim.x + threadIdx.x;

    // Bounds check
    if (TIdx >= arrayIdLength) return;

    // Each thread runs only if it sees a start flag of 1 (example condition)
    if (d_flagCellIdStart[TIdx] == 1 && d_flagCellIdEnd[TIdx] != 1) {

        // Outer loop over i
        for (int i = TIdx; i < arrayIdLength; i++) {
            // Check collisions with subsequent j
            for (int j = i + 1; j < arrayIdLength; j++) {
                // Perform collision check
                checkCollision(
                    i,
                    j, 
                    coordinates, 
                    d_objectIdArray, 
                    collDistSq, 
                    d_collisionCounter);

                // If flag says this is the last iteration for j, break afterward
                if (d_flagCellIdEnd[j] == 1) {
                    // One final iteration has just occurred, so exit inner loop
                    break;
                }
            }

            // If flag says this is the last iteration for i, break afterward
            if (d_flagCellIdEnd[i + 1] == 1) {
                break;
            }
        }
    }
}


int run(Coordinates h_vectorCoordinates, double collisionDistance) {
    unsigned int coordinatesLength = h_vectorCoordinates.size();
    unsigned int coordinatesSize = coordinatesLength * 3 * sizeof(double);  // since h_vectorCoordinates contained 3 double per row.
    unsigned int arrayIdLength = coordinatesLength * 8; // For every 3D coordinate get 1 cellId.
    unsigned int arrayIdSize = coordinatesLength* 8 * sizeof(int64_t);

    // Move the coordinate to device global memory.
    double *h_coordinates = vecToArr(h_vectorCoordinates);
    double *d_coordinates; CUDA_CHECK(hipMalloc(&d_coordinates, coordinatesSize));

    CUDA_CHECK(hipMemcpy(d_coordinates, h_coordinates, coordinatesSize, hipMemcpyHostToDevice));

    // Create cellIdArray and objectIdArray.
    int64_t *d_cellIdArray; CUDA_CHECK(hipMalloc(&d_cellIdArray, arrayIdSize));
    object_id *d_objectIdArray; CUDA_CHECK(hipMalloc(&d_objectIdArray, arrayIdLength * sizeof(object_id)));

    // Create array on device for out data from the Radix sort.        
    int64_t *d_cellIdArrayOut; CUDA_CHECK(hipMalloc(&d_cellIdArrayOut, arrayIdSize));
    object_id *d_objectIdArrayOut; CUDA_CHECK(hipMalloc(&d_objectIdArrayOut, arrayIdLength * sizeof(object_id)));

    // Take advantage of the fact that integer division drops the decimals
    unsigned int gridSize = coordinatesLength / MAX_BLOCK_SIZE;
    if (coordinatesLength % MAX_BLOCK_SIZE != 0) {
        gridSize += 1;
    }

    // Create the cellIdArray and the ObjectIdArray for the obtained coordinates.
    initializeObjectandCellArray<<<gridSize, MAX_BLOCK_SIZE>>>(
        d_coordinates, 
        d_cellIdArray, 
        d_objectIdArray, 
        coordinatesLength, 
        collisionDist);
    
    hipDeviceSynchronize();

    // Create a variable to hold the temporary storage.
    size_t temp_storage_bytes = 0;
    void *d_temp_storage = nullptr;

    // First call sets temp_storage_bytes
    hipcub::DeviceRadixSort::SortPairs(nullptr, temp_storage_bytes,
                                    d_cellIdArray, d_cellIdArrayOut,
                                    d_objectIdArray, d_objectIdArrayOut,
                                    arrayIdLength);

    CUDA_CHECK(hipMalloc(&d_temp_storage, temp_storage_bytes));

    // Second call does the actual sorting
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                    d_cellIdArray, d_cellIdArrayOut,
                                    d_objectIdArray, d_objectIdArrayOut,
                                    arrayIdLength);

    hipDeviceSynchronize();

    // Take advantage of the fact that integer division drops the decimals
    unsigned int gridSizeNmbCollisions = arrayIdLength / MAX_BLOCK_SIZE;
    if (coordinatesLength % MAX_BLOCK_SIZE != 0) {
        gridSizeNmbCollisions += 1;
    }

    // Create temporary arrays
    int64_t *d_cellIdArrayTemp = d_cellIdArray;
    object_id *d_objectIdArrayTemp = d_objectIdArray;

    // Create arrays on device that store information regarding when the cell id changes. 
    int *d_flagCellIdStart; CUDA_CHECK(hipMalloc(&d_flagCellIdStart, arrayIdLength * sizeof(int)));
    int *d_flagCellIdEnd; CUDA_CHECK(hipMalloc(&d_flagCellIdEnd, arrayIdLength * sizeof(int)));

    // Execute kernal that places start and end flags of the blocks of cell id in cellIdArray.
    placeCellIdFlags<<<gridSizeNmbCollisions, MAX_BLOCK_SIZE>>>(
        d_cellIdArrayOut, 
        d_cellIdArrayTemp, 
        d_objectIdArrayOut, 
        d_objectIdArrayTemp,
        d_flagCellIdStart,
        d_flagCellIdEnd,
        arrayIdLength);

    hipDeviceSynchronize();

    // Creates an collision counter array on the device, length equals the number of threads.
    int *d_collisionCounter; CUDA_CHECK(hipMalloc((&d_collisionCounter), arrayIdLength * sizeof(int)));
    CUDA_CHECK(hipMemset(d_collisionCounter, 0, arrayIdLength * sizeof(int)));
    double collDistSq = collisionDistance * collisionDistance;

    calculateNumberOfCollisions<<<gridSizeNmbCollisions, MAX_BLOCK_SIZE>>>(
        d_cellIdArrayOut,
        d_objectIdArrayOut, 
        d_flagCellIdStart,
        d_flagCellIdEnd,
        d_coordinates,
        collDistSq,
        arrayIdLength,
        d_collisionCounter);
        
    hipDeviceSynchronize();


    // Reducing the count array with built in cub function.
    int *d_collisionCounterOut = nullptr;
    CUDA_CHECK(hipMalloc((void**)&d_collisionCounterOut, sizeof(int)));

    // Resetting these so they don't clash with the earlier sort storage
    size_t temp_storage_bytes_reduce = 0;
    void *d_temp_storage_reduce = nullptr;

    // Checking for temporary storage
    hipcub::DeviceReduce::Sum(
        nullptr, 
        temp_storage_bytes_reduce,
        d_collisionCounter, 
        d_collisionCounterOut, 
        arrayIdLength);

    CUDA_CHECK(hipMalloc(&d_temp_storage_reduce, temp_storage_bytes_reduce));

    // Perform reduction
    hipcub::DeviceReduce::Sum(
        d_temp_storage_reduce, 
        temp_storage_bytes_reduce,
        d_collisionCounter,
        d_collisionCounterOut, 
        arrayIdLength);

    // Now copy single int to host
    int hostCollisionSum = 0;
    CUDA_CHECK(hipMemcpy(&hostCollisionSum, d_collisionCounterOut, 
                        sizeof(int), hipMemcpyDeviceToHost));

    // Freeing memory after operations.
    delete[] h_coordinates;

    hipFree(d_coordinates);
    
    hipFree(d_cellIdArray);
    hipFree(d_objectIdArray);
    hipFree(d_cellIdArrayOut);
    hipFree(d_objectIdArrayOut);

    hipFree(d_temp_storage);

    hipFree(d_flagCellIdStart);
    hipFree(d_flagCellIdEnd);

    hipFree(d_collisionCounter);
    hipFree(d_collisionCounterOut);
    hipFree(d_temp_storage_reduce);

    return hostCollisionSum;
}
